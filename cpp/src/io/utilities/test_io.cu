#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/utilities/pinned_host_vector.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/device_uvector.hpp>

#include <fcntl.h>
#include <fstream>
#include <iostream>
#include <sys/mman.h>
#include <unistd.h>

constexpr size_t data_size_bytes = 512 * 1024 * 1024;

void fill_random_data(int* data, size_t size)
{
  // Replace this function with your data generation logic if needed.
  for (size_t i = 0; i < size / sizeof(int); ++i) {
    data[i] = rand();
  }
}

void test_io(bool use_mmap, bool use_pinned)
{
  std::cout << "test_io: use_mmap=" << use_mmap << ", use_pinned=" << use_pinned << std::endl;

  std::string file_path = "datafile.bin";
  auto stream           = cudf::get_default_stream();

  std::vector<char> data;
  data.reserve(data_size_bytes);
  fill_random_data(reinterpret_cast<int*>(data.data()), data_size_bytes);

  std::ofstream file(file_path, std::ios::binary);
  file.write(data.data(), data_size_bytes);
  file.close();

  auto const fd = open(file_path.c_str(), O_RDONLY);
  CUDF_EXPECTS(fd != -1, "Error opening the file.");

  char const* h_data = nullptr;
  char* mapped_data  = nullptr;
  std::vector<char> read_data;
  if (use_mmap) {
    mapped_data = static_cast<char*>(mmap(nullptr, data_size_bytes, PROT_READ, MAP_PRIVATE, fd, 0));
    CUDF_EXPECTS(mapped_data != MAP_FAILED, "Error mapping the file into memory.");
    h_data = mapped_data;
  } else {
    read_data.resize(data_size_bytes);
    CUDF_EXPECTS(read(fd, read_data.data(), data_size_bytes) == data_size_bytes, "read failed");
    h_data = read_data.data();
  }

  // copy to GPU
  auto const chunk_size = 64ul * 1024 * 1024;
  cudf::detail::pinned_host_vector<char> pinned_data(use_pinned ? chunk_size : 0);

  rmm::device_uvector<char> d_data{chunk_size, stream};
  for (size_t chunk = 0; chunk < (data_size_bytes + chunk_size - 1) / chunk_size; ++chunk) {
    auto const offset = chunk * chunk_size;
    auto const size   = std::min(chunk_size, data_size_bytes - offset);

    auto src = h_data + offset;
    if (use_pinned) {
      std::memcpy(pinned_data.data(), src, size);
      src = pinned_data.data();
    }

    hipMemcpyAsync(d_data.data(), src, size, hipMemcpyDefault, stream);
    stream.synchronize();
  }

  close(fd);
  munmap(mapped_data, data_size_bytes);
}

void test_io_all_options()
{
  test_io(false, false);
  test_io(false, true);
  test_io(true, false);
  test_io(true, true);
}
